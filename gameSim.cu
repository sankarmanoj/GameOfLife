#include "constants.h"

// struct C2Iter
// {
//   int * data;
//   int width;
//   int position;
//   __device__ int operator[] (int n)
//   {
//     return data[position+n];
//   }
// };
//
// typedef struct C2Iter C2Iter;
//
// struct C1Iter
// {
//   int * data;
//   int width, height;
//   int position;
//   __device__ C2Iter operator[] (int n)
//   {
//     C2Iter x;
//     x.position = this->position + n*this->width;
//     return x;
//   }
// };
//
// typedef struct C1Iter C1Iter;
// struct Cube
// {
//   int * data;
//   int width,height,depth;
//   __device__  C1Iter operator[] (int n)
//   {
//     C1Iter x;
//     x.position = n*this->height*this->width;
//     x.data = this->data;
//     x.width = this->width;
//     x.height = this->height;
//     return x;
//   }
// };
// typedef struct Cube Cube;
static int * deviceFrame;
static hipDeviceProp_t properties;
extern void initDevice( int * initialFrame)
{
  hipMalloc((void **)&deviceFrame,sizeof(int)*TOTAL_POSITIONS);
  hipMemcpy(deviceFrame,initialFrame,sizeof(int)*TOTAL_POSITIONS,hipMemcpyHostToDevice);
  hipGetDeviceProperties(&properties,0);
}

extern void transformOperator(int * hostFrame)
{
  int area = properties.maxThreadsPerBlock / POSITION_WIDTH;

}
