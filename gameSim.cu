#include "hip/hip_runtime.h"
#include "constants.h"
struct cubearray
{
  int * data;
  int  position;
  __device__ __host__ int & operator[] (int3 index)
  {
    int cubeposition = index.z*POSITION_WIDTH*POSITION_HEIGHT + index.y*POSITION_WIDTH + index.z;
    return data[position+cubeposition];
  }
};
typedef struct cubearray cubearray;
 struct functor
{
  __device__ __host__  int operator() (cubearray input)
  {
    int count = input[make_int3(1,1,1)]+input[make_int3(1,1,0)]+input[make_int3(1,0,1)]+input[make_int3(0,1,1)];
    if(count>1)
    {
      return 1;
    }
    else{
      return 1;
    }
  }
};
typedef struct functor functor;
static int * deviceFrame;
extern void initDevice( int * initialFrame)
{
  hipMalloc((void **)&deviceFrame,sizeof(int)*TOTAL_POSITIONS);
  hipMemcpy(deviceFrame,initialFrame,sizeof(int)*TOTAL_POSITIONS,hipMemcpyHostToDevice);
}
__global__ void transformKernel(int * deviceFrame)
{
  int absolutePosition = (blockIdx.y*gridDim.x + blockIdx.x)*blockDim.x+ threadIdx.x;
  if(absolutePosition>TOTAL_POSITIONS)
  {
    return;
  }
  cubearray current_cube;
  current_cube.data = deviceFrame;
  current_cube.position = absolutePosition;
  functor x;
  current_cube[make_int3(1,1,1)] = x(current_cube);

}
extern  void transformOperator(int * hostFrame)
{
  int blocks = ceil((float)TOTAL_POSITIONS/1024);
  int xblocks=1,yblocks=1;
  if(blocks>65536)
  {
    xblocks=65536;
    yblocks=blocks/65536;
  }
  transformKernel<<<dim3(xblocks,yblocks),1024>>>(deviceFrame);
  hipDeviceSynchronize();
  hipMemcpy(hostFrame,deviceFrame,sizeof(int)*TOTAL_POSITIONS,hipMemcpyDeviceToHost);


}
