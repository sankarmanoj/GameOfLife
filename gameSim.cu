#include "hip/hip_runtime.h"
#include "constants.h"
#include <stdio.h>

__constant__ float weights[27];
struct cubearray
{
  int * data;
  int  position;
  __device__ __host__ int & operator[] (int3 index)
  {
    int cubeposition = index.z*4000 + index.y*200 + index.x;
    return data[position + cubeposition];
  }
};
typedef struct cubearray cubearray;
 class functor
{
public:
  functor(float * host_weights)
  {
    hipMemcpyToSymbol(HIP_SYMBOL(weights),host_weights,sizeof(float)*27);
  }
  __device__  int operator() (cubearray input)
  {
    int pos;
    float val = 0;
    for(int i = -1; i<1; i++)
      for(int j = -1; j<1; j++)
        for(int k = -1; k<1; k++)
        {

          val += weights[pos]*(float)input[make_int3(k,j,i)];
          pos++;
        }
    if(val>0.8)
    {
      return 1;
    }
    else{
      return 0;
    }

  }
};
typedef struct functor functor;
static int * deviceFrame;
static int * outputDeviceFrame;
extern void initDevice( int * initialFrame)
{
  hipMalloc((void **)&deviceFrame,sizeof(int)*TOTAL_POSITIONS);
  hipMemcpy(deviceFrame,initialFrame,sizeof(int)*TOTAL_POSITIONS,hipMemcpyHostToDevice);
  hipMalloc((void **)&outputDeviceFrame,sizeof(int)*TOTAL_POSITIONS);
  hipMemset(outputDeviceFrame,0,sizeof(int)*TOTAL_POSITIONS);

}
__global__ void transformKernel(int * deviceFrame , int * outputDeviceFrame, functor x)
{
  int absolutePosition = (blockIdx.y*gridDim.x + blockIdx.x)*1024+ threadIdx.x;

  if(absolutePosition>TOTAL_POSITIONS)
  {

    return;
  }
    // if(threadIdx.x<2)
    // printf("Absolute Position = %d , BlockIdx.(x,y) = (%d,%d) gridDim.x = %d\n",absolutePosition,blockIdx.x,blockIdx.y,gridDim.x);
  cubearray current_cube,output_cube;
  current_cube.data = deviceFrame;
  output_cube.data = outputDeviceFrame;
  output_cube.position = absolutePosition;
  current_cube.position = absolutePosition;
  output_cube[make_int3(0,0,0)]=x(current_cube);

}
extern  void transformOperator(int * hostFrame)
{
  int blocks = ceil((float)TOTAL_POSITIONS/1024);
  int xblocks=1,yblocks=1;
  if(blocks>65536)
  {
    xblocks=65536;
    yblocks=blocks/65536;
  }
  else{
    xblocks = blocks;
  }
  float hw [27] ;
  for(int i = 0; i<27;i++)
  {
    hw[i]=(float)rand()/RAND_MAX;
  }
  functor x(hw);
  // printf("Blocks = %d (x,y)=%d,%d\n",blocks,xblocks,yblocks);
  transformKernel<<<dim3(xblocks,yblocks),1024>>>(deviceFrame,outputDeviceFrame,x);
  hipDeviceSynchronize();
  hipMemcpy(hostFrame,outputDeviceFrame,sizeof(int)*TOTAL_POSITIONS,hipMemcpyDeviceToHost);
  hipMemcpyAsync(deviceFrame,outputDeviceFrame,sizeof(int)*TOTAL_POSITIONS,hipMemcpyDeviceToDevice);
  // memset(hostFrame,0,sizeof(int)*TOTAL_POSITIONS);

}
