#include "hip/hip_runtime.h"
#include "constants.h"
#include <stdio.h>
struct cubearray
{
  int * data;
  int  position;
  __device__ __host__ int & operator[] (int3 index)
  {
    int cubeposition = index.z*POSITION_WIDTH*POSITION_HEIGHT + index.y*POSITION_WIDTH + index.z;
    printf("%d\n", cubeposition+position);
    return data[position+cubeposition];
  }
};
typedef struct cubearray cubearray;
 struct functor
{
  __device__ __host__  int operator() (cubearray input)
  {
    int count = input[make_int3(1,1,1)]+input[make_int3(1,1,0)]+input[make_int3(1,0,1)]+input[make_int3(0,1,1)];
    if(count>1)
    {
      return 0;
    }
    else{
      return 0;
    }
  }
};
typedef struct functor functor;
static int * deviceFrame;
static int * outputDeviceFrame;
extern void initDevice( int * initialFrame)
{
  hipMalloc((void **)&deviceFrame,sizeof(int)*TOTAL_POSITIONS);
  hipMemcpy(deviceFrame,initialFrame,sizeof(int)*TOTAL_POSITIONS,hipMemcpyHostToDevice);
  hipMalloc((void **)&outputDeviceFrame,sizeof(int)*TOTAL_POSITIONS);
  hipMemcpy(outputDeviceFrame,initialFrame,sizeof(int)*TOTAL_POSITIONS,hipMemcpyHostToDevice);

}
__global__ void transformKernel(int * deviceFrame , int * outputDeviceFrame)
{
  int absolutePosition = (blockIdx.y*gridDim.x + blockIdx.x)*1024+ threadIdx.x;

  if(absolutePosition>TOTAL_POSITIONS)
  {

    return;
  }
    // if(threadIdx.x<2)
    // printf("Absolute Position = %d , BlockIdx.(x,y) = (%d,%d) gridDim.x = %d\n",absolutePosition,blockIdx.x,blockIdx.y,gridDim.x);
  cubearray current_cube,output_cube;
  current_cube.data = deviceFrame;
  output_cube.data = outputDeviceFrame;
  output_cube.position = absolutePosition;
  current_cube.position = absolutePosition;
  output_cube[make_int3(1,1,1)]=0;

}
extern  void transformOperator(int * hostFrame)
{
  int blocks = ceil((float)TOTAL_POSITIONS/1024);
  int xblocks=1,yblocks=1;
  if(blocks>65536)
  {
    xblocks=65536;
    yblocks=blocks/65536;
  }
  else{
    xblocks = blocks;
  }
  // printf("Blocks = %d (x,y)=%d,%d\n",blocks,xblocks,yblocks);
  transformKernel<<<dim3(1,1),1024>>>(deviceFrame,outputDeviceFrame);
  hipDeviceSynchronize();
  hipMemcpy(hostFrame,outputDeviceFrame,sizeof(int)*TOTAL_POSITIONS,hipMemcpyDeviceToHost);
  hipMemcpyAsync(deviceFrame,outputDeviceFrame,sizeof(int)*TOTAL_POSITIONS,hipMemcpyDeviceToDevice);

}
